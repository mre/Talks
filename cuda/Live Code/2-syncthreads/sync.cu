﻿
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 16

void print(unsigned int* data) {
  int i;
  for (i = 0; i < N; ++i) {
    printf("%d", data[i]);
  }
}
__global__ void kernel(unsigned int *dev_data)
{
	int x = threadIdx.x;
  dev_data[x] = 1;
/*	__syncthreads(); */
  //dev_data[x] += dev_data[N-x];
}

int main( void )
{
  unsigned int data[N] = {0};
	unsigned int *dev_data;
  size_t size = N*sizeof(unsigned int);
	hipMalloc((void **) &dev_data, size);
	hipMemcpy(dev_data, data, size, hipMemcpyHostToDevice);
	
	dim3 threads(16, 0);
	kernel<<<1, threads>>>(dev_data);

	hipMemcpy(data, dev_data, size, hipMemcpyDeviceToHost);
	hipFree(dev_data);
  print(data);
	return 0;
}
